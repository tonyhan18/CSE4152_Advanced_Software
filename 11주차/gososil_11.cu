#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

//1 prac 2
//2 prac 3
#define prac 1


#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;
#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

// CUDA event ��ü�� ����Ͽ� Ŀ�� ����ð� ����
void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL(hipEventDestroy(cuda_timer_start));
	CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

#define BLOCK_SIZE 32
#define MAX_N_ELEMENTS (1 << 20)

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

int N_EQU;
float* X0_GPU, * X1_GPU;
float* FX0_GPU, * FX1_GPU;
float* X0, * X1;
float* FX0, * FX1;
float* A, * B, * C;


hipError_t find_roots_GPU(float* A, float* B, float* C, float* X0_GPU, float* X1_GPU, float* FX0_GPU, float* FX1_GPU);

__global__ void find_roots_Kernel(float* d_A, float* d_B, float* d_C, float* d_X0, float* d_X1, float* d_FX0, float* d_FX1)
{

    int id;
    float a, b, c, d, x0, x1, temp;

    id = blockDim.x * blockIdx.x + threadIdx.x;
    a = d_A[id];
    b = d_B[id];
    c = d_C[id];
    d = sqrtf(b * b - 4.0f * a * c);
    temp = 1.0f / (2.0f * a);
    d_X0[id] = x0 = (-b - d) * temp;
    d_X1[id] = x1 = (-b + d) * temp;
    d_FX0[id] = (a * x0 + b) * x0 + c;
    d_FX1[id] = (a * x1 + b) * x1 + c;

}

void find_roots_CPU(float* A, float* B, float* C, float* X0, float* X1, float* FX0, float* FX1, int n) {
    int j;
    float a, b, c, d, x0, x1, temp;

    for (j = 0; j < n; ++j)
    {
        a = A[j];
        b = B[j];
        c = C[j];
        d = sqrtf(b * b - 4.0f * a * c);
        temp = 1.0f / (2.0f * a);
        X0[j] = x0 = (-b - d) * temp;
        X1[j] = x1 = (-b + d) * temp;
        FX0[j] = (a * x0 + b) * x0 + c;
        FX1[j] = (a * x1 + b) * x1 + c;
    }
}

void init_bin_file(int size) {
    FILE* fp = fopen("A.bin", "wb");
    int random_num;
    
    srand((unsigned)time(NULL));
    printf(">>> Binary File init Started<<<\n");
    fwrite(&size, sizeof(int), 1, fp);
    
    for (int i = 0; i < size; ++i)
    {
        random_num = (float)((float)rand() / RAND_MAX * 20.0f - 10.0f);
        fwrite(&random_num, sizeof(float), 1, fp);
    }
    fclose(fp);

    fp = fopen("B.bin", "wb");
    fwrite(&size, sizeof(int), 1, fp);

    for (int i = 0; i < size; ++i)
    {
        random_num = (float)((float)rand() / RAND_MAX * 1000.0f + 100.0f);
        fwrite(&random_num, sizeof(float), 1, fp);
    }
    fclose(fp);

    fp = fopen("C.bin", "wb");
    fwrite(&size, sizeof(int), 1, fp);

    for (int i = 0; i < size; ++i)
    {
        random_num = (float)((float)rand() / RAND_MAX * 20.0f - 10.0f);
        fwrite(&random_num, sizeof(float), 1, fp);
    }

    fclose(fp);
    printf(">>> Binary File init End <<\n\n");
}

void read_bin_file()
{
    FILE* fp = fopen("A.bin", "rb");
    /*Todo*/

    printf(">>> Binary File Read Started <<<\n");
    fread(&N_EQU, sizeof(int), 1, fp);

    A = new float[N_EQU];
    B = new float[N_EQU];
    C = new float[N_EQU];
    X0 = new float[N_EQU];
    X1 = new float[N_EQU];
    FX0 = new float[N_EQU];
    FX1 = new float[N_EQU];
    X0_GPU = new float[N_EQU];
    X1_GPU = new float[N_EQU];
    FX0_GPU = new float[N_EQU];
    FX1_GPU = new float[N_EQU];


    fread(A, sizeof(float), N_EQU, fp);

    fclose(fp);

    fp = fopen("B.bin", "rb");
    fread(&N_EQU, sizeof(int), 1, fp);
    fread(B, sizeof(float), 1, fp);
    fclose(fp);

    fp = fopen("C.bin", "rb");
    fread(&N_EQU, sizeof(int), 1, fp);
    fread(C, sizeof(float), 1, fp);
    fclose(fp);

    printf(">>> Binary File Read End <<<\n\n");
}

int main()
{
    int i;
    FILE* fp;
    init_bin_file(MAX_N_ELEMENTS);
    read_bin_file();

    printf("N_EQUATION : %d.\n", N_EQU);

    //CPU 
    printf(">>> find_roots_CPU Started <<<\n");
    CHECK_TIME_START;
    find_roots_CPU(A, B, C, X0, X1, FX0, FX1, N_EQU);
    CHECK_TIME_END(compute_time);
    printf(">>> find_roots_CPU End <<<\n\n");
    printf("CPU time = %.6f\n", compute_time);

    //GPU
    printf(">>> find_roots_GPU Started <<<\n");
    find_roots_GPU(A, B, C, X0_GPU, X1_GPU, FX0_GPU, FX1_GPU);
    printf(">>> find_roots_GPU End <<<\n\n");
    printf("GPU time = %.6f\n", device_time);


    fp = fopen("X0.bin", "wb");
    if (!fp)
    {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }

    fwrite(&N_EQU, sizeof(int), 1, fp);
    for (i = 0; i < N_EQU; ++i)
        fwrite(&X0_GPU[i], sizeof(float), 1, fp);
    fclose(fp);


    fp = fopen("X1.bin", "wb");
    if (!fp)
    {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }

    fwrite(&N_EQU, sizeof(int), 1, fp);

    for (i = 0; i < N_EQU; ++i)
        fwrite(&X1_GPU[i], sizeof(float), 1, fp);
    fclose(fp);


    fp = fopen("FX0.bin", "wb");
    if (!fp)
    {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }
    fwrite(&N_EQU, sizeof(int), 1, fp);

    for (i = 0; i < N_EQU; ++i)
        fwrite(&FX0_GPU[i], sizeof(float), 1, fp);
    fclose(fp);
    printf(">>> end <<<\n\n");


    fp = fopen("FX1.bin", "wb");
    if (!fp)
    {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }

    fwrite(&N_EQU, sizeof(int), 1, fp);
    for (i = 0; i < N_EQU; ++i)
        fwrite(&X0_GPU[i], sizeof(float), 1, fp);

    fclose(fp);

    return (0);
}


hipError_t find_roots_GPU(float* A, float* B, float* C, float* X0_GPU, float* X1_GPU, float* FX0_GPU, float* FX1_GPU)
{
    CHECK_TIME_INIT_GPU();

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    /*Todo*/

    float* d_A, * d_B, * d_C;
    float* d_X, * d_Y;
    float* d_FX, * d_FY;

    size_t size;

    size = N_EQU * sizeof(float);

    CUDA_CALL(hipMalloc(&d_A, size))

        CUDA_CALL(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice))

        CUDA_CALL(hipMalloc(&d_B, size))

        CUDA_CALL(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice))

        CUDA_CALL(hipMalloc(&d_C, size))

        CUDA_CALL(hipMemcpy(d_C, C, size, hipMemcpyHostToDevice))


        CUDA_CALL(hipMalloc(&d_X, size))
        CUDA_CALL(hipMalloc(&d_Y, size))
        CUDA_CALL(hipMalloc(&d_FX, size))
        CUDA_CALL(hipMalloc(&d_FY, size))

        size = N_EQU;

    CHECK_TIME_START_GPU()

        dim3 dimBlock(BLOCK_SIZE, 1);
    dim3 dimGrid(N_EQU / dimBlock.x, 1);

    find_roots_Kernel << < dimGrid, dimBlock >> > (d_A, d_B, d_C, d_X, d_Y, d_FX, d_FY);

    CHECK_TIME_END_GPU(device_time)
        size = N_EQU * sizeof(float);

    CUDA_CALL(hipGetLastError())

        CUDA_CALL(hipDeviceSynchronize())
        CUDA_CALL(hipMemcpy(X0_GPU, d_X, size, hipMemcpyDeviceToHost))

        CUDA_CALL(hipMemcpy(X1_GPU, d_Y, size, hipMemcpyDeviceToHost))

        CUDA_CALL(hipMemcpy(FX0_GPU, d_FX, size, hipMemcpyDeviceToHost))

        CUDA_CALL(hipMemcpy(FX1_GPU, d_FY, size, hipMemcpyDeviceToHost))


        CHECK_TIME_DEST_GPU();

Error:
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_X);
    hipFree(d_Y);
    hipFree(d_FX);
    hipFree(d_FY);

    return cudaStatus;
}